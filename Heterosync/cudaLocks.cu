#include "cudaLocks.h"

hipError_t cudaLocksInit(const int maxBlocksPerKernel, const int numMutexes,
                          const int numSemaphores,
                          const bool pageAlign, /* const region_t locksReg*/
                          const int NUM_SM)
{
  hipError_t cudaErr = hipGetLastError();
  checkError(cudaErr, "Start cudaLocksInit");

  hipHostMalloc(&cpuLockData, sizeof(cudaLockData_t));

  if (maxBlocksPerKernel <= 0)    return hipErrorNotInitialized;
  if (numMutexes <= 0)            return hipErrorNotInitialized;
  if (numSemaphores <= 0)         return hipErrorNotInitialized;

  // initialize some of the lock data's values
  cpuLockData->maxBufferSize          = maxBlocksPerKernel;
  cpuLockData->arrayStride            = (maxBlocksPerKernel + NUM_SM) / 16 * 16;
  cpuLockData->mutexCount             = numMutexes;
  cpuLockData->semaphoreCount         = numSemaphores;

  hipMalloc(&cpuLockData->barrierBuffers,   sizeof(unsigned int) * cpuLockData->arrayStride * 2);

  hipMalloc(&cpuLockData->mutexBuffers,     sizeof(int) * cpuLockData->arrayStride * cpuLockData->mutexCount);
  hipMalloc(&cpuLockData->mutexBufferHeads, sizeof(unsigned int) * cpuLockData->mutexCount);
  hipMalloc(&cpuLockData->mutexBufferTails, sizeof(unsigned int) * cpuLockData->mutexCount);

  hipMalloc(&cpuLockData->semaphoreBuffers, sizeof(unsigned int) * 5 * cpuLockData->semaphoreCount);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);

  cudaErr = hipGetLastError();
  checkError(cudaErr, "Before memset");

  hipDeviceSynchronize();
  hipEventRecord(start, 0);

  hipMemset(cpuLockData->barrierBuffers, 0, sizeof(unsigned int) * cpuLockData->arrayStride * 2);

  hipMemset(cpuLockData->mutexBufferHeads, 0, sizeof(unsigned int) * cpuLockData->mutexCount);
  hipMemset(cpuLockData->mutexBufferTails, 0, sizeof(unsigned int) * cpuLockData->mutexCount);

  // initialize mutexBuffers to appropriate values
  // initialize to -1 to ensure that sleep mutex doesn't accidentally read the
  // wrong TB ID
  //for (int j = 0; j < mutexCount; ++j) {
  for (int i = 0; i < (cpuLockData->arrayStride * cpuLockData->mutexCount); ++i) {
    // set the first location for each SM to 1 so that the ring buffer can be
    // used by the first TB right away (otherwise livelock because no locations
    // ever == 1)
    if (i % cpuLockData->arrayStride == 0) {
      hipMemset(&(cpuLockData->mutexBuffers[i]), 1, sizeof(int));
    }
    // for all other locations initialize to -1 so TBs for these locations
    // don't think it's their turn right away
    else {
      // ** TODO: Could copy a whole bunch of these over at once to reduce number of memsets
      hipMemset(&(cpuLockData->mutexBuffers[i]), -1, sizeof(int));
    }
  }
  /*
  for (int i = 0; i < cpuLockData->mutexCount; ++i) {
    // set the first location for each SM to 1 so that the ring buffer can be
    // used by the first TB right away (otherwise livelock because no locations
    // ever == 1)
    hipMemset(&(cpuLockData->mutexBuffers[i]), 1, sizeof(int));
    hipMemset(&(cpuLockData->mutexBuffers[i]), -1, (cpuLockData->arrayStride - 1) * sizeof(int));
  }
  */

  hipMemset(cpuLockData->semaphoreBuffers, 0, sizeof(unsigned int) * cpuLockData->semaphoreCount * 5);

  hipDeviceSynchronize();
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float elapsedTime = 0.0f;
  hipEventElapsedTime(&elapsedTime, start, end);
  fprintf(stdout, "\tmemcpy H->D 1 elapsed time: %f ms\n", elapsedTime);
  fflush(stdout);

  hipEventDestroy(start);
  hipEventDestroy(end);

  return hipSuccess;
}

hipError_t cudaLocksDestroy()
{
  if (cpuLockData == NULL) { return hipErrorNotInitialized; }
  hipFree(cpuLockData->barrierBuffers);
  hipFree(cpuLockData->mutexBuffers);
  hipFree(cpuLockData->mutexBufferHeads);
  hipFree(cpuLockData->mutexBufferTails);
  hipFree(cpuLockData->semaphoreBuffers);

  hipHostFree(cpuLockData);

  return hipSuccess;
}
